#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "stdio.h"

__global__ void kernel_calculate (particule * tab, particule_pos * tab_ret) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < NbData) {
		int j = 0;
		float acX = 0;
		float acY = 0;
		float acZ = 0;
		//printf("Dans Boucle 1 \n");
		for ( j = 0; j < NbData; j++ ) {
			float tempo_acc = 0; 
			temp tempo;
			//printf("Boucle 2 numéro %d \n", j);
			if (j != i) {	
		
				// Parrallel 1
				tempo.deltaX = tab[j].pos.posX - tab[i].pos.posX;
				tempo.deltaY = tab[j].pos.posY - tab[i].pos.posY;
				tempo.deltaZ = tab[j].pos.posZ - tab[i].pos.posZ;
				// Fin parrallel 1

				tempo.d_ij = sqrtf( tempo.deltaX*tempo.deltaX + tempo.deltaY*tempo.deltaY + tempo.deltaZ*tempo.deltaZ );

				if (tempo.d_ij < 1)
					tempo.d_ij = 1;

				tempo_acc = MASS_FACTOR*DAMPING_FACTOR*(1/(tempo.d_ij*tempo.d_ij*tempo.d_ij))*tab[j].masse; // Meme calcul realise trois fois

				// Parrallel 2
				acX += tempo.deltaX * tempo_acc;
				acY += tempo.deltaY * tempo_acc;
				acZ += tempo.deltaZ * tempo_acc;
				// Fin Parrallel 2

			}
					
		}
			//tab[i].acc.accX = acX;
			//tab[i].acc.accY = acY;
			//tab[i].acc.accZ = acZ;
			
			// Parrallel 0
			tab[i].mob.mobX += acX;
			tab[i].pos.posX += tab[i].mob.mobX * 0.1f;
			tab_ret[i].posX = tab[i].pos.posX; 

			tab[i].mob.mobY += acY;
			tab[i].pos.posY += tab[i].mob.mobY * 0.1f;
			tab_ret[i].posY = tab[i].pos.posY; 

			tab[i].mob.mobZ += acZ;
			tab[i].pos.posZ += tab[i].mob.mobZ * 0.1f;
			tab_ret[i].posZ = tab[i].pos.posZ; 
			// Fin

	}

	return;
}

void CalculateMove_k (int NbBlock, int NbThread, particule * tab, particule_pos * tab_ret) {
	kernel_calculate<<<NbBlock,NbThread>>>(tab, tab_ret);
	return;
}
