#include "hip/hip_runtime.h"

__global__ void kernel_saxpy( int n, float a, float * x, float * y, float * z ) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n ) { 
		z[i] = a * x[i] + y [i];
	}
}

void saxpy( int nblocks, int nthreads, int n, float a, float * x, float * y, float * z ) {
	kernel_saxpy<<<nblocks, nthreads>>>( n, a, x, y, z );
}
